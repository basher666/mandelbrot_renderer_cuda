#include <hip/hip_runtime.h>

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelbrot_kernel(int *canvas, int *num_it, double l_margin, double r_margin, double u_margin, double d_margin, int N)
{
	int num_rows = blockDim.y*gridDim.y;
	int num_cols = blockDim.x*gridDim.x;
	double z_n_x = 0;
	double z_n_y = 0;
	double tmp_x, tmp_y;
	int tid_x = blockIdx.x*blockDim.x + threadIdx.x;
	int tid_y = blockIdx.y*blockDim.y + threadIdx.y;
	double c_x = l_margin + (tid_x/(double)(num_cols -1))*(r_margin - l_margin);
	double c_y = d_margin + (tid_y/(double)(num_rows -1))*(u_margin - d_margin);
	int escape_time = 0;
	int idx = tid_y*num_cols + tid_x;

	while(z_n_x*z_n_x + z_n_y*z_n_y < 4 && escape_time<N)
	{
		tmp_x = z_n_x*z_n_x - z_n_y*z_n_y;
		tmp_y = 2*z_n_x*z_n_y;
		z_n_x = tmp_x + c_x;
		z_n_y = tmp_y + c_y;
		escape_time ++;
	}
	if (escape_time==N)
		canvas[idx] = 0;
	else
	{
		double mod = z_n_x*z_n_x + z_n_y*z_n_y;
		canvas[idx] = (int)(((escape_time - log(log(mod))/log(2.0))/(double)N)*255.0);
		// canvas[idx] = (int)(((double)escape_time/(double) N)*255.0);
	}
	num_it[idx] = escape_time;
}

void render(int *h_canvas,long double center_x,long double center_y, double init_len, int dim_x, int dim_y)
{
	hipError_t err = hipSuccess;
	
	double l_margin = center_x - init_len/2.0;
	double r_margin = center_x + init_len;
	double u_margin = center_y + init_len/2.0;
	double d_margin = center_y - init_len/2.0;
	int N = 255;
	dim3 threads_per_block(32,32,1);
	dim3 blocks_per_grid(dim_x/32,dim_y/32,1);
	
	size_t canvas_size =  dim_x*dim_y*sizeof(int);

	

	int *h_num_it = (int*)malloc(canvas_size);
	memset(h_num_it, 0, canvas_size);

	int *d_canvas = NULL;
	err = hipMalloc((void **)&d_canvas, canvas_size);
	if(err != hipSuccess)
	{
		printf("Error in hipMalloc : d_canvas\n");
		exit(EXIT_FAILURE);
	}

	int *d_num_it = NULL;
	err = hipMalloc((void **)&d_num_it, canvas_size);
	if(err != hipSuccess)
	{
		printf("Error in hipMalloc : d_num_it\n");
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_canvas, h_canvas, canvas_size, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		printf("Error in hipMemcpy : d_canvas\n");
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_num_it, h_num_it, canvas_size, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		printf("Error in hipMemcpy : d_num_it\n");
		exit(EXIT_FAILURE);
	}

	mandelbrot_kernel <<<blocks_per_grid, threads_per_block>>> (d_canvas, d_num_it, l_margin, r_margin, u_margin, d_margin, N);

	err = hipGetLastError();
	if(err!=hipSuccess)
	{
		printf("Error in kernel\n");
		exit(EXIT_FAILURE);
	}

	// printf("Getting the canvas back from kernel\n");
	// fflush(stdout);

	err = hipMemcpy(h_canvas, d_canvas, canvas_size, hipMemcpyDeviceToHost);
	if(err!=hipSuccess)
	{
		printf("Error in cudaMemcpy: h_canvas\n");
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(h_num_it, d_num_it, canvas_size, hipMemcpyDeviceToHost);
	if(err!=hipSuccess)
	{
		printf("Error in cudaMemcpy: h_num_it\n");
		exit(EXIT_FAILURE);
	}

	// printf("Freeing device memory\n");
	// fflush(stdout);
	err = hipFree(d_canvas);
	if(err!=hipSuccess)
	{
		printf("Error in cudaFree: d_canvas\n");
		exit(EXIT_FAILURE);
	}

	err = hipDeviceReset();
	if(err!=hipSuccess)
	{
		printf("Error in cudaDeviceReset\n");
		exit(EXIT_FAILURE);
	}	

	// printf("analyzing escape times\n");
	// fflush(stdout);
	
	// int max_esc = 0;
	// int min_esc = 1000;
	// double avg_esc = 0.0;
	// int outside_count = 0;
	// for(int i=0;i<dim_y;i++)
	// {
	// 	for(int j=0;j<dim_x;j++)
	// 	{
	// 		if (h_canvas[i*dim_x + j]!=0)
	// 		{
	// 			if (h_num_it[i*dim_x + j]>max_esc)
	// 				max_esc = h_num_it[i*dim_x + j];
	// 			if (h_num_it[i*dim_x + j]<min_esc)
	// 				min_esc = h_num_it[i*dim_x + j];
	// 			avg_esc += h_num_it[i*dim_x + j];
	// 			outside_count ++;
	// 		}
	// 	}
	// }
	// avg_esc = avg_esc/outside_count;
	// printf("max it:%d, min_it:%d, outside_count:%d, avg_it:%lf",max_esc, min_esc, outside_count, avg_esc);


	
}